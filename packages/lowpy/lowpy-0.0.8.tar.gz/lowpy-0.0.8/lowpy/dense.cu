
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void propagate(
    const   int                     I,
    const   int                     iteration,
    const   double * __restrict__   x,
    const   double * __restrict__   w,
    const   double * __restrict__   b,
            double * __restrict__   y,
            double * __restrict__   z
){
    int j = blockIdx.x;
    double sum = 0;
    int inputIdx = 0;
    if (iteration > -1){
        inputIdx = iteration * I;
    }
    for (int i = 0; i < I; i++) sum += x[inputIdx+i]*w[j*I+i];
    y[j] = sum + b[j];
    z[j] = 1/(1 + exp(-y[j]));
}
__global__ void backpropagate(
    const   int                     iteration,
    const   int                     label,
            double * __restrict__   dedz,
    const   double * __restrict__   z,
    const   int                     J_n,
            double * __restrict__   w_n,
    const   double * __restrict__   dedz_n,
    const   double * __restrict__   dzdy_n,
            double * __restrict__   dzdy,
    const   double                  alpha,
    const   int                     I,
            double * __restrict__   b,
            double * __restrict__   w,
    const   double * __restrict__   x,
    const   double                  beta,
            double * __restrict__   vtb,
            double * __restrict__   vtw
){
    int j   = blockIdx.x;
    int I_n = gridDim.x;
    int inputIdx = 0;
    if (iteration > -1){
        inputIdx = iteration * I;
    }
    if (label > -1){
        if (j == label){
            dedz[j] = z[j] - 1;
        }else{
            dedz[j] = z[j] - 0;
        }
    }else{
        double sum = 0;
        for (int j_n = 0; j_n < J_n; j_n++) sum += w_n[j+j_n*I_n] * dedz_n[j_n] * dzdy_n[j_n];
        dedz[j] = sum;
    }
    dzdy[j] = z[j] * (1 - z[j]);
    b[j]   -= (beta * vtb[j] + alpha * dedz[j] * dzdy[j]);
    vtb[j]  = beta * vtb[j] + alpha * dedz[j] * dzdy[j];

    for (int i = 0; i < I; i++){
        w[j*I+i]    -= (beta * vtw[j*I+i] + alpha * dedz[j] * dzdy[j] * x[inputIdx+i]);
        vtw[j*I+i]  = beta * vtw[j*I+i] + alpha * dedz[j] * dzdy[j] * x[inputIdx+i];
    }
}
__global__ void argmax(
        const   int                     label,
        const   double * __restrict__   z,
                double *__restrict__    hits
){
    int j = blockIdx.x;
    int J = gridDim.x;
    if (j == 0){
        double maxVal = 0;
        int maxIdx = 0;
        for (int i = 0; i < J; i++){
            if (z[i] > maxVal){
                maxIdx = i;
                maxVal = z[i];
            }
        }
        if (maxIdx == label){
            hits[0] += 1;
        }
        hits[1] = maxIdx;
        hits[2] = maxVal;
    }
}